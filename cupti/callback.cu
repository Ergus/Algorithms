#include "hip/hip_runtime.h"
// Copyright (C) 2024  Jimmy Aguilar Mena

// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.

// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.	 See the
// GNU General Public License for more details.

// You should have received a copy of the GNU General Public License
// along with this program.	 If not, see <http://www.gnu.org/licenses/>.

#pragma nv_diag_suppress 815 // suppress consr return warning
#include <argparser.hpp>

#include <vector>
#include <random>
#include <algorithm>

#include "utils.h"


/**
   Example kernel r[] = a1[] + a2[]
*/
__global__ void kernelExample(size_t size, const float *a1, const float *a2, float *r)
{
	int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (globalIdx < size)
		r[globalIdx] = a1[globalIdx] + a2[globalIdx];
}


/**
   Wrapper function for kernel r[] = a1[] + a2[]
*/
std::vector<float> functionExample(
	const std::vector<float> &a1,
	const std::vector<float> &a2
) {
	const size_t size = a1.size();
	myassert(a2.size() ==  size);

	float *d_a1, *d_a2, *d_r;
	hipMalloc((void**)&d_a1, size * sizeof(float));
	hipMalloc((void**)&d_a2, size * sizeof(float));
	hipMalloc((void**)&d_r, size * sizeof(float));

	hipMemcpy(d_a1, a1.data(), size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_a2, a2.data(), size * sizeof(float), hipMemcpyHostToDevice);

	constexpr size_t blockdim = 128;
	size_t nblocks = (size + blockdim - 1) / blockdim;

	kernelExample<<<nblocks, blockdim>>>(size, d_a1, d_a2, d_r);

	hipFree(d_a2);
	hipFree(d_a1);

	std::vector<float> r(size);
	hipMemcpy(r.data(), &d_r[0], size * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_r);

	return r;
}


int main(int argc, char **argv)
{
	argparser::init(argc, argv);
	const size_t size = argparser::cl<int>("array_size");

	std::vector<float> a1(size);
	std::vector<float> a2(size);

	static std::random_device rd;	 // you only need to initialize it once
	static std::mt19937 mte(rd());	 // this is a relative big object to create

	std::uniform_int_distribution<int> dist(0, 1024); // dist(mte)

	std::generate(a1.begin(), a1.end(), [&dist](){ return dist(mte); });
	std::generate(a2.begin(), a2.end(), [&dist](){ return dist(mte); });

	std::vector<float> result = functionExample(a1, a2);

	// In this case the float sum in gpu is exactly the same.
	for (size_t i = 0; i < result.size(); ++i)
		myassert(a1[i] + a2[i] == result[i]);

	return 0;
}
